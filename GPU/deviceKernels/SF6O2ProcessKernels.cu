#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <context.hpp>

#define rho_Si 5.02e7 // in (1e15 atoms/cm³) (rho Si)
#define k_sigma_Si 300. // in (1e15 cm⁻²s⁻¹)
#define beta_sigma_Si 5.0e-2 // in (1e15 cm⁻²s⁻¹)
#define MASK_MAT 0

extern "C" __global__ void calculateEtchRate(const NumericType *rates,
                                             const NumericType *coverages,
                                             const NumericType *materialIds,
                                             NumericType *etchRate,
                                             const unsigned long numPoints,
                                             const NumericType totalIonFlux,
                                             const NumericType totalEtchantFlux,
                                             const NumericType totalOxygenFlux)
{
  unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int stride = blockDim.x * gridDim.x;

  const NumericType *ionSputteringRate = rates;
  const NumericType *ionEnhancedRate = rates + numPoints;
  const NumericType *eCoverage = coverages;

  for (; tidx < numPoints; tidx += stride)
  {
    if ((int)materialIds[tidx] != MASK_MAT)
    {
      // 1e4 converts the rates to micrometers/s
      etchRate[tidx] = -(1. / rho_Si) * 1e4 *
                       (k_sigma_Si * eCoverage[tidx] / 4. +
                        ionSputteringRate[tidx] * totalIonFlux +
                        eCoverage[tidx] * ionEnhancedRate[tidx] * totalIonFlux);
    }
    else
    {
      etchRate[tidx] = 0.;
    }
  }
}

extern "C" __global__ void updateCoverages(const NumericType *rates,
                                           NumericType *coverages,
                                           const unsigned long numPoints,
                                           const NumericType totalIonFlux,
                                           const NumericType totalEtchantFlux,
                                           const NumericType totalOxygenFlux)
{
  unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int stride = blockDim.x * gridDim.x;

  const NumericType *ionEnhancedRate = rates + numPoints;
  const NumericType *oxygenSputteringRate = rates + 2 * numPoints;
  const NumericType *etchantRate = rates + 3 * numPoints;
  const NumericType *oxygenRate = rates + 4 * numPoints;

  NumericType *eCoverage = coverages;
  NumericType *oCoverage = coverages + numPoints;

  for (; tidx < numPoints; tidx += stride)
  {
    if (etchantRate[tidx] < 1e-6)
    {
      eCoverage[tidx] = 0;
    }
    else
    {
      eCoverage[tidx] =
          etchantRate[tidx] * totalEtchantFlux /
          (etchantRate[tidx] * totalEtchantFlux +
           (k_sigma_Si + 2 * ionEnhancedRate[tidx] * totalIonFlux) *
               (1 + (oxygenRate[tidx] * totalOxygenFlux) /
                        (beta_sigma_Si +
                         oxygenSputteringRate[tidx] * totalIonFlux)));
    }

    if (oxygenRate[tidx] < 1e-6)
    {
      oCoverage[tidx] = 0;
    }
    else
    {
      oCoverage[tidx] =
          oxygenRate[tidx] * totalOxygenFlux /
          (oxygenRate[tidx] * totalOxygenFlux +
           (beta_sigma_Si + oxygenSputteringRate[tidx] * totalIonFlux) *
               (1 +
                (etchantRate[tidx] * totalEtchantFlux) /
                    (k_sigma_Si + 2 * ionEnhancedRate[tidx] * totalIonFlux)));
    }
  }
}