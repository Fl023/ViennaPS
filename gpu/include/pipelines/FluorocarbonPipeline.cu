#include "hip/hip_runtime.h"
#include <optix_device.h>

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <raygBoundary.hpp>
#include <raygLaunchParams.hpp>
#include <raygPerRayData.hpp>
#include <raygRNG.hpp>
#include <raygReflection.hpp>
#include <raygSBTRecords.hpp>
#include <raygSource.hpp>

#include <models/psPlasmaEtchingParameters.hpp>

#include <vcContext.hpp>
#include <vcVectorType.hpp>

using namespace viennaray::gpu;

extern "C" __constant__ LaunchParams launchParams;
enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };

/* --------------- ION --------------- */

extern "C" __global__ void __closesthit__FCIon() {
  const HitSBTData *sbtData = (const HitSBTData *)optixGetSbtDataPointer();
  PerRayData *prd = (PerRayData *)getPRD<PerRayData>();

  if (sbtData->isBoundary) {
    if (params.periodicBoundary) {
      applyPeriodicBoundary(prd, sbtData);
    } else {
      reflectFromBoundary(prd);
    }
  } else {
    auto primID = optixGetPrimitiveIndex();
    int material = launchParams.materialIds[primID];
    viennaps::FluorocarbonParameters<float> *params =
        reinterpret_cast<viennaps::FluorocarbonParameters<float> *>(
            launchParams.customData);

    auto geomNormal = computeNormal(sbtData, primID);
    auto cosTheta = -viennacore::DotProduct(prd->dir, geomNormal);
    float angle = acosf(max(min(cosTheta, 1.f), 0.f));

    const float sqrtE = sqrtf(prd->energy);
    const float f_e_sp = (1 + B_sp * (1 - cosTheta * cosTheta)) * cosTheta;
    const float Y_sp = Ae_sp * max(sqrtE - sqrt_Eth_sp, 0.f) * f_e_sp;
    const float Y_ie = Ae_ie * max(sqrtE - sqrt_Eth_ie, 0.f) * cosTheta;
    const float Y_p = Ap_ie * max(sqrtE - sqrt_Eth_p, 0.f) * cosTheta;

    // sputtering yield Y_sp ionSputteringFlux
    atomicAdd(&params.resultBuffer[getIdx(0, &params)], Y_sp);

    // ion enhanced etching yield Y_ie ionEnhancedFlux
    atomicAdd(&params.resultBuffer[getIdx(1, &params)], Y_ie);

    // ion enhanced O sputtering yield Y_O ionPolymerFlux
    atomicAdd(&params.resultBuffer[getIdx(2, &params)], Y_p);

    // ---------- REFLECTION ------------ //
    // Small incident angles are reflected with the energy fraction centered at
    // 0
    float Eref_peak = 0.f;
    float A = 1. / (1. + params->Ions.n_l *
                             (M_PI_2f / params->Ions.inflectAngle - 1.));
    if (angle >= params->Ions.inflectAngle) {
      Eref_peak = 1 - (1 - A) * (M_PI_2f - angle) /
                          (M_PI_2f - params->Ions.inflectAngle);
    } else {
      Eref_peak = A * pow(angle / params->Ions.inflectAngle, params->Ions.n_l);
    }

    // Gaussian distribution around the Eref_peak scaled by the particle energy
    float newEnergy;
    do {
      newEnergy = getNormalDistRand(&prd->RNGstate) * prd->energy * 0.1f +
                  Eref_peak * prd->energy;
    } while (newEnergy > prd->energy || newEnergy <= 0.f);

    // Set the flag to stop tracing if the energy is below the threshold
    float minEnergy = min(params->Substrate.Eth_ie, params->Substrate.Eth_sp);
    if (newEnergy > minEnergy) {
      prd->energy = newEnergy;
      conedCosineReflection(prd, geomNormal,
                            M_PI_2f - min(angle, params->Ions.minAngle));
    } else {
      prd->energy = -1.f;
    }
  }
}

extern "C" __global__ void __miss__FCIon() {
  getPRD<PerRayData>()->rayWeight = 0.f;
}

extern "C" __global__ void __raygen__FCIon() {
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dims = optixGetLaunchDimensions();
  const int linearLaunchIndex =
      idx.x + idx.y * dims.x + idx.z * dims.x * dims.y;

  // per-ray data
  PerRayData prd;
  // each ray has its own RNG state
  initializeRNGState(&prd, linearLaunchIndex, launchParams.seed);

  // initialize ray position and direction
  initializeRayPosition(&prd, &launchParams);
  initializeRayDirection(&prd, launchParams.cosineExponent);

  viennaps::PlasmaEtchingParameters<float> *params =
      reinterpret_cast<viennaps::PlasmaEtchingParameters<float> *>(
          launchParams.customData);
  float minEnergy = min(params->Substrate.Eth_ie, params->Substrate.Eth_sp);
  do {
    prd.energy = getNormalDistRand(&prd.RNGstate) * params->Ions.sigmaEnergy +
                 params->Ions.meanEnergy;
  } while (prd.energy < minEnergy);

  // the values we store the PRD pointer in:
  uint32_t u0, u1;
  packPointer((void *)&prd, u0, u1);

  while (continueRay(launchParams, prd)) {
    optixTrace(launchParams.traversable, // traversable GAS
               make_float3(prd.pos[0], prd.pos[1], prd.pos[2]), // origin
               make_float3(prd.dir[0], prd.dir[1], prd.dir[2]), // direction
               1e-4f,                                           // tmin
               1e20f,                                           // tmax
               0.0f,                                            // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,              // SBT offset
               RAY_TYPE_COUNT,                // SBT stride
               SURFACE_RAY_TYPE,              // missSBTIndex
               u0, u1);
  }
}

/* --------------- ETCHANT --------------- */

extern "C" __global__ void __closesthit__FCEtchant() {
  const HitSBTData *sbtData = (const HitSBTData *)optixGetSbtDataPointer();
  PerRayData *prd = (PerRayData *)getPRD<PerRayData>();

  if (sbtData->isBoundary) {
    if (launchParams.periodicBoundary) {
      applyPeriodicBoundary(prd, sbtData);
    } else {
      reflectFromBoundary(prd);
    }
  } else {
    atomicAdd(&launchParams.resultBuffer[getIdx(0, &launchParams)],
              prd->rayWeight);

    // ------------- REFLECTION --------------- //
    const unsigned int primID = optixGetPrimitiveIndex();
    float *data = (float *)sbtData->cellData;
    const float &phi_e = data[primID];
    int material = launchParams.materialIds[primID];

    /// TODO:
    // Check material ID
    float gamma_e = launchParams.materialSticking[material];
    const float Seff = gamma_e * max(1.f - phi_e, 0.f);
    prd->rayWeight -= prd->rayWeight * Seff;
    diffuseReflection(prd);
  }
}

extern "C" __global__ void __miss__FCEtchant() {
  getPRD<PerRayData>()->rayWeight = 0.f;
}

extern "C" __global__ void __raygen__FCEtchant() {
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dims = optixGetLaunchDimensions();
  const int linearLaunchIndex =
      idx.x + idx.y * dims.x + idx.z * dims.x * dims.y;

  // per-ray data
  PerRayData prd;
  // each ray has its own RNG state
  initializeRNGState(&prd, linearLaunchIndex, launchParams.seed);

  // initialize ray position and direction
  initializeRayPosition(&prd, &launchParams);
  initializeRayDirection(&prd, launchParams.cosineExponent);

  // the values we store the PRD pointer in:
  uint32_t u0, u1;
  packPointer((void *)&prd, u0, u1);

  while (continueRay(launchParams, prd)) {
    optixTrace(launchParams.traversable, // traversable GAS
               make_float3(prd.pos[0], prd.pos[1], prd.pos[2]), // origin
               make_float3(prd.dir[0], prd.dir[1], prd.dir[2]), // direction
               1e-4f,                                           // tmin
               1e20f,                                           // tmax
               0.0f,                                            // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,              // SBT offset
               RAY_TYPE_COUNT,                // SBT stride
               SURFACE_RAY_TYPE,              // missSBTIndex
               u0, u1);
  }
}

/* ------------- POLYMER --------------- */

extern "C" __global__ void __closesthit__FCPolymer() {
  const HitSBTData *sbtData = (const HitSBTData *)optixGetSbtDataPointer();
  PerRayData *prd = (PerRayData *)getPRD<PerRayData>();

  if (sbtData->isBoundary) {
    if (launchParams.periodicBoundary) {
      applyPeriodicBoundary(prd, sbtData);
    } else {
      reflectFromBoundary(prd);
    }
  } else {
    atomicAdd(&launchParams.resultBuffer[getIdx(0, &launchParams)],
              prd->rayWeight);

    // ------------- REFLECTION --------------- //
    const unsigned int primID = optixGetPrimitiveIndex();
    float *data = (float *)sbtData->cellData;
    const float &phi_e = data[primID];
    const float &phi_p = data[primID + launchParams.numElements];
    int material = launchParams.materialIds[primID];

    float gamma_pe = launchParams.materialSticking[material];
    const float Seff = gamma_pe * max(1.f - phi_e - phi_p, 0.f);
    prd->rayWeight -= prd->rayWeight * Seff;
    diffuseReflection(prd);
  }
}

extern "C" __global__ void __miss__FCPolymer() {
  getPRD<PerRayData>()->rayWeight = 0.f;
}

extern "C" __global__ void __raygen__FCPolymer() {
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dims = optixGetLaunchDimensions();
  const int linearLaunchIndex =
      idx.x + idx.y * dims.x + idx.z * dims.x * dims.y;

  // per-ray data
  PerRayData prd;
  // each ray has its own RNG state
  initializeRNGState(&prd, linearLaunchIndex, launchParams.seed);

  // initialize ray position and direction
  initializeRayPosition(&prd, &launchParams);
  initializeRayDirection(&prd, launchParams.cosineExponent);

  // the values we store the PRD pointer in:
  uint32_t u0, u1;
  packPointer((void *)&prd, u0, u1);

  while (continueRay(launchParams, prd)) {
    optixTrace(launchParams.traversable, // traversable GAS
               make_float3(prd.pos[0], prd.pos[1], prd.pos[2]), // origin
               make_float3(prd.dir[0], prd.dir[1], prd.dir[2]), // direction
               1e-4f,                                           // tmin
               1e20f,                                           // tmax
               0.0f,                                            // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,              // SBT offset
               RAY_TYPE_COUNT,                // SBT stride
               SURFACE_RAY_TYPE,              // missSBTIndex
               u0, u1);
  }
}
